#include "hip/hip_runtime.h"
//	author: Benedykt Bela

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <time.h>

using namespace std;



//	ilosc zmiennych typu double, ktore mozemy przechowywac w pamieci GPU
const int rozmiar = 220000000;	// 262135808
//	tablicowa wartosc stalej pi sluzaca do wyliczania bledu
const double pi = 3.1415926535897932;



//	funcja na GPU sluzaca do wypelniania tablicy liczbami ze wzoru Leibniza dla typu float
__global__ void dodawanie_float_fill(float *tab, int rozmiar)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x + 1;

	//	jezeli indeks i jest mniejszy od rozmiaru wypelnianej tablicy
	if (i <= rozmiar)
		tab[i - 1] = 4 * pow(-1, i - 1) / (2 * i - 1);
}



//	funkcja na GPU sluzaca do dodawania rownoleglego liczb typu float z tablicy -
//	w miejce tablicy o indeksie i zapisujemy wartosc sumy liczby spod tego indeksu 
//	oraz liczby spod indeksu i + krok
//	i - wartosci od zera do polowy wielkosci zadanego wektora
//	krok - suma i oraz zmiennej krok daje indeks oddalony o polowe dlugosci wektora
__global__ void dodawanie_float(float *tab, int half, int krok)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	//	jezeli indeks i jest mniejszy od polowy wektora skladnikow
	if (i < half)
		tab[i] = tab[i] + tab[i + krok];
}



//	ponizsze dwie funkcje sa analogiczne jak powyzsze dwie, ale operuja na zmiennych typu double
__global__ void dodawanie_double_fill(double *tab, int rozmiar)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
	if (i <= rozmiar)
		tab[i - 1] = 4 * pow(-1, i - 1) / (2 * i - 1);
}



__global__ void dodawanie_double(double *tab, int half, int krok)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < half)
		tab[i] = tab[i] + tab[i + krok];
}



//	deklaracje funkcji zdefiniowanych w dalszej czesci
void CPU_float();
void CPU_double();
void GPU_float();
void GPU_double();



int main()
{
	cout << endl;

	
	//	zmienne potrzebne do obliczania czasu wykonania danego fragmentu programu
	double start, stop;
	
	//	zmiennej start nadajemy wartosc czasu w momencie rozpoczecia programu
	start = clock();

	//	wykonania programu na GPU oraz zmiennych typu float
	GPU_float();

	//	zmiennej stop nadajemy wartosc czasu po wykonaniu funkcji GPU_float()
	stop = clock();
	//	roznica stop - start daje nam czas wykonania programu co wypisujemy w konsoli
	cout << "Czas wykonania na GPU float:   " << stop - start << endl << endl;


	//	ponizszy kod dziala analogicznie jak ten powyzej z ta ronica, ze wykonuje sie odpowiednio
	//	na GPU dla typu double oraz na CPU dla typu float oraz na CPU dla typu double
	start = clock();

	GPU_double();

	stop = clock();
	cout << "Czas wykonania na GPU double:   " << stop - start << endl << endl;


	start = clock();

	CPU_float();

	stop = clock();
	cout << "Czas wykonania na CPU float:   " << stop - start << endl << endl;


	start = clock();

	CPU_double();

	stop = clock();
	cout << "Czas wykonania na CPU double:   " << stop - start << endl << endl;
	

	cout << endl << endl;
    return 0;
}



//	program do obliczania liczby pi na GPU dla typu float
void GPU_float()
{
	//	tworze wskaznik na tablice zmiennych typu float dla CPU oraz GPU
	float *d_tab = new float[rozmiar];
	float *tab = new float[rozmiar];
	//	alokuje potrzebna pamiec na GPU
	hipMalloc(&d_tab, rozmiar * sizeof(float));
	//	zmienna blocks mowi nam ile blokow watkow po 1024 watki musimy odpalic zeby wykonac dane zadanie
	int blocks = rozmiar / 1024 + 1;

	//	wypelniam tablice danymi zgodnie ze wzorem Leibniza na GPU
	dodawanie_float_fill << <blocks, 1024 >> > (d_tab, rozmiar);

	//	actual_size zawiera aktualny rozmiar sumowanej tablicy
	int actual_size = rozmiar;
	//	zmienna half zawiera rozmiar polowy sumowanej tablicy zaokraglajac w dol
	int half = actual_size / 2;
	//	zmienna krok zawiera rozmiar polowy sumowanej tablicy zaokraglajac w gore
	int krok = (actual_size + 1) / 2;


	//	wykonuje petle dopoki rozmiar sumowanej tablicy jest wiekszy niz 1
	while (actual_size > 1)
	{
		//	ponizsza konstrukcja if-else zapisuje do zmiennej blocks polowe jej poprzedniej wartosci
		//	zaokraglajac w gore
		if (blocks % 2 == 0)
			blocks = blocks / 2;
		else
			blocks = blocks / 2 + 1;

		//	sumuje po dwa skladniki umieszczajac obliczona sume w miejscu pierwszego skladnika
		dodawanie_float << <blocks, 1024 >> > (d_tab, half, krok);

		//	ponizsza konstrukcja if-else zapisuje aktualne wielkosci sumowanej tablicy
		if (actual_size % 2 == 0)
			actual_size = actual_size / 2;
		else
			actual_size = (actual_size / 2) + 1;

		//	aktualizuje ponizsze zmienne zgodnie z zasada opisana przy deklaracji
		half = actual_size / 2;
		krok = (actual_size + 1) / 2;
	}
	

	//	kopiuje z GPU sume wszystkich skladnikow sumowanego wektora, ktora znajduje sie 
	//	w tym momencie pod adresem pierszego elementu tablicy d_tab
	hipMemcpy(&tab[0], &d_tab[0], sizeof(float), hipMemcpyDeviceToHost);


	//	wypisuje wyliczona wartosc liczby pi oraz blad obliczony zgodnie z opisem w instrukcji do laboratorium
	cout <<"GPU float:   "<<setprecision(20)<< tab[0] << endl;
	cout <<"GPU float blad:   "<<setprecision(16)<< ((double)tab[0]-pi)/pi << endl;

	//	zwalniam miejsce wykorzystywane w tej funkcji na GPU
	hipFree(d_tab);
}



//	analogicznie jak GPU_float, ale dla zmiennych typu double
void GPU_double()
{
	double *d_tab = new double[rozmiar];
	double *tab = new double[rozmiar];
	hipMalloc(&d_tab, rozmiar * sizeof(double));
	int blocks = rozmiar / 1024 + 1;

	dodawanie_double_fill << <blocks, 1024 >> > (d_tab, rozmiar);

	int actual_size = rozmiar;
	int half = actual_size / 2;
	int krok = (actual_size + 1) / 2;


	while (actual_size > 1)
	{
		if (blocks % 2 == 0)
			blocks = blocks / 2;
		else
			blocks = blocks / 2 + 1;

		dodawanie_double << <blocks, 1024 >> > (d_tab, half, krok);

		if (actual_size % 2 == 0)
			actual_size = actual_size / 2;
		else
			actual_size = (actual_size / 2) + 1;

		half = actual_size / 2;
		krok = (actual_size + 1) / 2;
	}


	hipMemcpy(&tab[0], &d_tab[0], sizeof(double), hipMemcpyDeviceToHost);


	cout << "GPU double:   " << setprecision(20) << tab[0] << endl;
	cout << "GPU double blad:   " << setprecision(16) << (tab[0] - pi) / pi << endl;

	hipFree(d_tab);
}


//	funkcja wykonuje dokladnie to samo co jej odpowiednik na GPU, tylko zamiast obliczen rownoleglych
//	zastosowana jest petla for wykonujaca obliczenie jedno po drugim
void CPU_float() 
{
	float *abc = new float[rozmiar];


	for (int i = 1; i <= rozmiar; i++)
	{
		abc[i-1] = 4 * pow(-1, i - 1) / (2 * i - 1);
	}


	int actual_size = rozmiar;
	int half = actual_size / 2;
	int krok = (actual_size + 1) / 2;

	
	while (actual_size > 1)
	{
		for (int i = 0; i < half; i++)
			abc[i] = abc[i] + abc[i + krok];

		if (actual_size % 2 == 0)
			actual_size = actual_size / 2;
		else
			actual_size = (actual_size / 2) + 1;

		half = actual_size / 2;
		krok = (actual_size + 1) / 2;
	}


	cout << setprecision(20) << "CPU float:    " << abc[0] << endl;
	cout << "CPU float blad:   " << setprecision(16) << ((double)abc[0] - pi) / pi << endl;
}



//	analogicznie jak CPU_float, ale dla zmiennych typu double
void CPU_double()
{
	double *abc = new double[rozmiar];


	for (int i = 1; i <= rozmiar; i++)
	{
		abc[i - 1] = 4 * pow(-1, i - 1) / (2 * i - 1);
	}
	

	int actual_size = rozmiar;
	int half = actual_size / 2;
	int krok = (actual_size + 1) / 2;


	while (actual_size > 1)
	{
		for (int i = 0; i < half; i++)
			abc[i] = abc[i] + abc[i + krok];

		if (actual_size % 2 == 0)
			actual_size = actual_size / 2;
		else
			actual_size = (actual_size / 2) + 1;

		half = actual_size / 2;
		krok = (actual_size + 1) / 2;
	}


	cout << setprecision(20) << "CPU double:    " << abc[0] << endl;
	cout << "CPU double blad:   " << setprecision(16) << (abc[0] - pi) / pi << endl;
}


